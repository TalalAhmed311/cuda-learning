#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



// For CPU
void vector_add_cpu(float *a, float *b, float *c,int n){

    for(int i=0;i<n;i++){
        c[i] = a[i]+b[i];
    }

}

// For GPU
__global__ void vector_add_gpu(float *a,float *b, float *c, int n){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx<n){
        c[idx]  = a[idx] + b[idx];
    }

}

void init_vector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}


int main(int argc, char const *argv[])
{
    int N = 10000000;
    int BLOCK_SIZE = 256;
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu;
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c_cpu = (float*)malloc(size);
    h_c_gpu = (float*)malloc(size);

    // Initialize vectors
    srand(time(NULL));
    init_vector(h_a, N);
    init_vector(h_b, N);

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    

    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

        printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        vector_add_gpu<<<num_blocks, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    // Print results
    printf("CPU average time: %f milliseconds\n", cpu_avg_time*1000);
    printf("GPU average time: %f milliseconds\n", gpu_avg_time*1000);
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);


    // Free memory
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
